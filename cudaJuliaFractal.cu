#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <stdint.h>

extern "C" {
  #include "png_util.h"
}

#define T 16

/**
 * Uses kernel to iterate through image pixels.
 * Calculates the color for each pixel from the equation for a Julia Fractal.
 */
__global__ void cudaDrawKernel(double cR, double cI, int size, int iter, int64_t *c_julia) {

  double newZR, newZI;
  int x,y;

  x = (threadIdx.x + blockDim.x*blockIdx.x);
  y = (threadIdx.y + blockDim.y*blockIdx.y);

  if (x<size && y<size) {
    newZR = 1.5*(x-size*0.5)/(size*0.5);
    newZI = (y-size*0.5)/(size*0.5);

    double oldZR, oldZI, color;
    int i, count = 0;

    for (i = 0; i < iter; i++) {
      oldZR = newZR;
      oldZI = newZI;

      newZR = oldZR*oldZR - oldZI*oldZI + cR;
      newZI = (2.f)*oldZR*oldZI + cI;
    
      if ((newZR*newZR + newZI*newZI) <= 4.0) {
	count++;
      }
    }

    color = sqrt((double)count);
    c_julia[y+x*size] = color;
  }
}

/**
 * Sets image variables.
 * Starts Cuda kernel.
 * Creates Image from data.
 */
int main(int argc, char** argv) {
  
  int size = atoi(argv[1]);
  int iter = atoi(argv[2]);
  double cR = -0.778;
  double cI = -0.116;
  
  
  /* Start Cuda Time */

  hipEvent_t tic, toc;
  hipEventCreate(&tic);
  hipEventCreate(&toc);
  hipEventRecord(tic, 0);
  
  
  /* Allocate and Copy Memory for Cuda */

  int64_t *h_julia = (int64_t*) calloc(size*size, sizeof(int64_t));
  int64_t *c_julia;
  hipMalloc(&c_julia, size*size*sizeof(int64_t));
  hipMemcpy(c_julia, h_julia, size*size*sizeof(int64_t), hipMemcpyHostToDevice);

  
  /* Run the kernel */
  
  int g = (size+T-1)/T;
  dim3 gDim(g, g);
  dim3 bDim(T, T);
  cudaDrawKernel <<< gDim, bDim >>> (cR, cI, size, iter, c_julia);

  
  /* Copy Memory back from Cuda */
  
  hipMemcpy(h_julia, c_julia, size*size*sizeof(int64_t), hipMemcpyDeviceToHost);

  
  /* End Cuda Time */
  
  hipEventRecord(toc, 0);
  hipEventSynchronize(toc);
  float elapsed;
  hipEventElapsedTime(&elapsed, tic, toc);
  printf("Elapsed time: %g\n", elapsed/1000.0);

  
  /* Image Creation */
  
  double timeA = clock();

  FILE *png = fopen("CudaJuliaFractal.png", "w");
  write_hot_png(png, size, size, h_julia, 0, 80);
  fclose(png);

  double timeB = clock();
  double elapsedPic = (timeB-timeA)/CLOCKS_PER_SEC;
  printf("Image creation time: %f\n", elapsedPic);

  return 0;
}
